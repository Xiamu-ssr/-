#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define INDENT "    "
#define MAX_THREADS 40960
#define THREADS_PER_VECTOR 4
#define MAX_THREADS_PER_BLOCK 256
#define VECTORS_PER_BLOCK (MAX_THREADS_PER_BLOCK/THREADS_PER_VECTOR)
#define BLOCKS_PER_GRID (MAX_THREADS/MAX_THREADS_PER_BLOCK)
const char* version_name = "naive base-line";\

void preprocess(dist_matrix_t *mat, data_t *x, data_t *y) {
}

void destroy_additional_info(void *additional_info) {
}


template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

template <typename IndexType, typename ValueType>
__global__ void My_spmv_csr_kernel(const IndexType row_num,
                       const IndexType * A_row_offset,
                       const IndexType * A_col_index,
                       const ValueType * A_value,
                       const ValueType * x,
                       ValueType * y)
{
    const IndexType THREADS_PER_BLOCK = VECTORS_PER_BLOCK * THREADS_PER_VECTOR;
    const IndexType thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;    // global thread index
    const IndexType thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);          // thread index within the vector
    const IndexType row_id   = thread_id   /  THREADS_PER_VECTOR;               // global vector index

    if(row_id < row_num){
        const IndexType row_start = A_row_offset[row_id];                  //same as: row_start = Ap[row];
        const IndexType row_end   = A_row_offset[row_id+1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for(IndexType jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * x[ A_col_index[jj] ];

        sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
        if (thread_lane == 0){
            y[row_id] = sum;
        }   
    }
}

inline int ceiling(int num, int den) {
    return (num - 1) / den + 1;
}

void spmv(dist_matrix_t *mat, const data_t* x, data_t* y) {
    // int *row_counter;
    // hipMalloc(&row_counter, sizeof(int));
    // hipMemset(row_counter, 0, sizeof(int));
    int m = mat->global_m;
    // dim3 grid_size (ceiling(m, blockDimx), 1, 1);
    const unsigned int NUM_BLOCKS = static_cast<unsigned int>((m + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);

    My_spmv_csr_kernel<index_t, data_t><<<NUM_BLOCKS, MAX_THREADS_PER_BLOCK>>>(m, mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, x, y);
    // hipFree(row_counter);
}
